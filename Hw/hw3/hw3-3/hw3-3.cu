#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>

using namespace std;


//======================
#define DEV_NO 0
#define BLOCK_SIZE 32
hipDeviceProp_t prop;

const int INF = ((1 << 30) - 1);
int n, m, n_padded;
unsigned int* host_dist_s;
unsigned int* host_dist_t;
unsigned int* device_dist;

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    // pad array size to multiples of 32
    n_padded = ((n + 31) / 32) * 32;
    host_dist_s = (unsigned int*) malloc(n_padded * n_padded * sizeof(unsigned int));

    for (int i = 0; i < n_padded; ++i) {
        for (int j = 0; j < n_padded; ++j) {
            if (i < n && j < n && i == j) {
                host_dist_s[i * n_padded + j] = 0;
            } else {
                host_dist_s[i * n_padded + j] = INF;
            }
        }
    }

    int* edges = (int*)malloc(m * 3 * sizeof(int));
    fread(edges, sizeof(int), m * 3, file);
    for (int i = 0; i < m; ++i) {
        host_dist_s[edges[i * 3 + 0] * n_padded + edges[i * 3 + 1]] = edges[i * 3 + 2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (host_dist_t[i * n_padded + j] >= INF) host_dist_t[i * n_padded + j] = INF;
        }
        fwrite(&host_dist_t[i * n_padded], sizeof(unsigned int), n, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void calKernelPhase1(unsigned int* Dist, int n, int B, int Round, int block_start_y, int block_start_x){

    // block index
    int block_x = block_start_x + blockIdx.x;
    int block_y = block_start_y + blockIdx.y;

    // thread index
    int x = threadIdx.x;
    int y = threadIdx.y;

    // global index
    int i = block_y * B + y;
    int j = block_x * B + x;

    // move to share memory
    __shared__ unsigned int shared_dist[BLOCK_SIZE][BLOCK_SIZE];
    shared_dist[y][x] = Dist[i * n + j];

    __syncthreads();

    // For each block, it need to compute B times
    for (int k = 0; k < BLOCK_SIZE; ++k) { // each phase will perform B iterations
        shared_dist[y][x] = min(shared_dist[y][x], shared_dist[y][k] + shared_dist[k][x]);
        __syncthreads();
    }

    // move back to GPU memory
    Dist[i * n + j] = shared_dist[y][x];
}


__global__ void calKernelPhase2(unsigned int* Dist, int n, int row_col, int B, int Round, int block_start_y, int block_start_x){
    
    // block index
    int block_x = block_start_x + blockIdx.x;
    int block_y = block_start_y + blockIdx.y;

    // thread index
    int x = threadIdx.x;
    int y = threadIdx.y;

    // global index
    int i = block_y * B + y;
    int j = block_x * B + x;

    // pivot index
    int pivot_start = Round * B;

    __shared__ unsigned int shared_pivot[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ unsigned int shared_dist[BLOCK_SIZE][BLOCK_SIZE];

    // use registers to cache the points to be calculated
    unsigned int vertex = Dist[i * n + j];

    // init shared memory
    shared_dist[y][x] = vertex;
    if(row_col == 0)
        shared_pivot[y][x] = Dist[i * n + (pivot_start + x)];
    else if(row_col == 1)
        shared_pivot[y][x] = Dist[(pivot_start + y) * n + j];


    __syncthreads();

    // Computation
    if(row_col == 0){ // pivot row
        // For each block, it need to compute B times
        #pragma unroll 32
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            vertex = min(vertex, shared_pivot[y][k] + shared_dist[k][x]);
        }
    }
    else if(row_col == 1){ // pivot col
        // For each block, it need to compute B times
        #pragma unroll 32
        for (int k = 0; k < BLOCK_SIZE; ++k) { 
            vertex = min(vertex, shared_dist[y][k] + shared_pivot[k][x]);
        }
    }

    // write back to GPU memory
    Dist[i * n + j] = vertex;
}


__global__ void calKernelPhase3(unsigned int* Dist, int n, int B, int Round, int block_start_y, int block_start_x){

    // block index
    int block_x = block_start_x + blockIdx.x;
    int block_y = block_start_y + blockIdx.y;

    // thread index
    int x = threadIdx.x;
    int y = threadIdx.y;

    // global index
    int i = block_y * B + y;
    int j = block_x * B + x;

    // pivot index
    int pivot_start = Round * B;

    __shared__ unsigned int shared_row[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ unsigned int shared_col[BLOCK_SIZE][BLOCK_SIZE];

    // use registers to cache the points to be calculated
    unsigned int vertex = Dist[i * n + j];

    // init shared memory
    shared_row[y][x] = Dist[i * n + (pivot_start + x)];
    shared_col[y][x] = Dist[(pivot_start + y) * n + j];

    __syncthreads();

    // Computation
    // For each block, it need to compute B times
    #pragma unroll 32
    for (int k = 0; k < BLOCK_SIZE; ++k) {
        vertex = min(vertex, shared_row[y][k] + shared_col[k][x]);
    }
    
    // write back to GPU memory
    Dist[i * n + j] = vertex;
}

void calPhase1(int n, int B, int Round, int block_start_y, int block_start_x){
    dim3 blockSize(B, B);
    dim3 gridSize(1, 1);
    calKernelPhase1<<<gridSize, blockSize>>>(device_dist, n, B, Round, block_start_y, block_start_x);
}

void calPhase2(int n, int row_col, int B, int Round, int block_start_y, int block_start_x, int height_num_blocks, int width_num_blocks){
    dim3 blockSize(B, B);
    dim3 gridSize(width_num_blocks, height_num_blocks);
    calKernelPhase2<<<gridSize, blockSize>>>(device_dist, n, row_col, B, Round, block_start_y, block_start_x);
}

void calPhase3(int n, int B, int Round, int block_start_y, int block_start_x, int height_num_blocks, int width_num_blocks){
    dim3 blockSize(B, B);
    dim3 gridSize(width_num_blocks, height_num_blocks);
    calKernelPhase3<<<gridSize, blockSize>>>(device_dist, n, B, Round, block_start_y, block_start_x);
}

void block_FW(int B) {
    int round = ceil(n_padded, B);
    for (int r = 0; r < round; ++r) {
        /* Phase 1*/
        // pivot block
        calPhase1(n_padded, B, r, r, r);

        /* Phase 2*/
        calPhase2(n_padded, 0, B, r, r, 0, 1, r); // pivot row: from 0 to now index
        calPhase2(n_padded, 0, B, r, r, r + 1, 1, round - r - 1); // pivot row: from now index + 1 to end
        calPhase2(n_padded, 1, B, r, 0, r, r, 1);  // pivot col: from 0 to now index
        calPhase2(n_padded, 1, B, r, r + 1, r, round - r - 1, 1); // pivot col: from now index + 1 to end

        /* Phase 3*/
        calPhase3(n_padded, B, r, 0, 0, r, r); // other: left upper
        calPhase3(n_padded, B, r, 0, r + 1, r, round - r - 1); // other: right upper
        calPhase3(n_padded, B, r, r + 1, 0, round - r - 1, r); // other: left lower
        calPhase3(n_padded, B, r, r + 1, r + 1, round - r - 1, round - r - 1); // ohter: right lower
    }
}


int main(int argc, char* argv[]) {
    input(argv[1]);
    host_dist_t = (unsigned int*) malloc(n_padded * n_padded * sizeof(unsigned int));

    hipGetDeviceProperties(&prop, DEV_NO);
    printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    
    // cuda malloc memory
    hipMalloc((void**)&device_dist, n_padded * n_padded * sizeof(unsigned int));

    // copy host memory to cuda
    hipMemcpy(device_dist, host_dist_s, n_padded * n_padded * sizeof(unsigned int), hipMemcpyHostToDevice);

    // Blocked Floyd-Warshall
    block_FW(BLOCK_SIZE);

    // copy cuda memory back to host
    hipMemcpy(host_dist_t, device_dist, n_padded * n_padded * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // output
    output(argv[2]);

    // free memory
    free(host_dist_s);
    free(host_dist_t);
    hipFree(device_dist);

    return 0;
}