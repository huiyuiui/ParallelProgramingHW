#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


//======================
#define DEV_NO 0
#define BLOCK_SIZE 32
hipDeviceProp_t prop;

const int INF = ((1 << 30) - 1);
int n, m;
unsigned int* host_dist_s;
unsigned int* host_dist_t;
unsigned int* device_dist;

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    host_dist_s = (unsigned int*) malloc(n * n * sizeof(unsigned int));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                host_dist_s[i * n + j] = 0;
            } else {
                host_dist_s[i * n + j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        host_dist_s[pair[0] * n + pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (host_dist_t[i * n + j] >= INF) host_dist_t[i * n + j] = INF;
        }
        fwrite(&host_dist_t[i * n], sizeof(unsigned int), n, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }

__global__ void calKernel(unsigned int* Dist, int n, int B, int Round, int block_start_y, int block_start_x) {

    // block index
    int block_x = block_start_x + blockIdx.x;
    int block_y = block_start_y + blockIdx.y;

    // global index
    int i = block_y * B + threadIdx.y;
    int j = block_x * B + threadIdx.x;

    if(i >= n || j >= n) return;

    // init pivot index
    int pivot_start = Round * B;
    int pivot_end = min((Round + 1) * B, n);

    // For each block, it need to compute B times
    for (int r = 0; r < pivot_end - pivot_start; ++r) { // each phase will perform B iterations
        int k = pivot_start + r;
        if(Dist[i * n + k] + Dist[k * n + j] < Dist[i * n + j])
            Dist[i * n + j] = Dist[i * n + k] + Dist[k * n + j];
        __syncthreads();
    }
}

void cal(int n, int B, int Round, int block_start_y, int block_start_x, int height_num_blocks, int width_num_blocks){
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(width_num_blocks, height_num_blocks);
    calKernel<<<gridSize, blockSize>>>(device_dist, n, B, Round, block_start_y, block_start_x);
}

void block_FW(int B) {
    int round = ceil(n, B);
    for (int r = 0; r < round; ++r) {
        /* Phase 1*/
        // pivot block
        dim3 blockSize(B, B);
        dim3 gridSize(1, 1);
        calKernel<<<gridSize, blockSize>>>(device_dist, n, B, r, r, r);

        /* Phase 2*/
        cal(n, B, r, r, 0, 1, r); // pivot row: from 0 to now index
        cal(n, B, r, r, r + 1, 1, round - r - 1); // pivot row: from now index + 1 to end
        cal(n, B, r, 0, r, r, 1);  // pivot col: from 0 to now index
        cal(n, B, r, r + 1, r, round - r - 1, 1); // pivot col: from now index + 1 to end

        /* Phase 3*/
        cal(n, B, r, 0, 0, r, r); // other: left upper
        cal(n, B, r, 0, r + 1, r, round - r - 1); // other: right upper
        cal(n, B, r, r + 1, 0, round - r - 1, r); // other: left lower
        cal(n, B, r, r + 1, r + 1, round - r - 1, round - r - 1); // ohter: right lower
    }
}


int main(int argc, char* argv[]) {
    input(argv[1]);
    host_dist_t = (unsigned int*) malloc(n * n * sizeof(unsigned int));

    hipGetDeviceProperties(&prop, DEV_NO);
    printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    
    // cuda malloc memory
    hipMalloc((void**)&device_dist, n * n * sizeof(unsigned int));

    // copy host memory to cuda
    hipMemcpy(device_dist, host_dist_s, n * n * sizeof(unsigned int), hipMemcpyHostToDevice);

    // Blocked Floyd-Warshall
    block_FW(BLOCK_SIZE);

    // copy cuda memory back to host
    hipMemcpy(host_dist_t, device_dist, n * n * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // output
    output(argv[2]);

    // free memory
    free(host_dist_s);
    free(host_dist_t);
    hipFree(device_dist);

    return 0;
}