#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace std;

//======================
#define DEV_NO 0
#define BLOCK_SIZE 32
hipDeviceProp_t prop;

int B, N, d;
float *Q, *K, *V, *O;
float *dev_Q, *dev_K, *dev_V, *dev_O;

void input(char *input_filename);

void output(char *output_filename);

double getTimeStamp() {
    struct timeval tv;
    gettimeofday( &tv, NULL );
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

__global__ void flash_attention_kernel(float* q, float* k, float* v, float* o, float* l, float* m, int d);

int main(int argc, char* argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input_filename> <output_filename>\n", argv[0]);
        return 1;
    }

    input(argv[1]);

    hipGetDeviceProperties(&prop, DEV_NO);
    printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    
    // cuda malloc memory
    hipMalloc((void**)&dev_Q, B * N * d * sizeof(float));
    hipMalloc((void**)&dev_K, B * N * d * sizeof(float));
    hipMalloc((void**)&dev_V, B * N * d * sizeof(float));
    hipMalloc((void**)&dev_O, B * N * d * sizeof(float));

    // copy host memory to cuda
    hipMemcpy(dev_Q, Q, B * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_K, K, B * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_V, V, B * N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_O, O, B * N * d * sizeof(float), hipMemcpyHostToDevice);

    // kernel parameters
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blockNum(N / BLOCK_SIZE, N / BLOCK_SIZE);

    double start, end;
    start = getTimeStamp();

    // flash attention
    for (int i = 0; i < B; i++){
        // init l and m
        float *l = (float *)malloc(N * sizeof(float));
        float *m = (float *)malloc(N * sizeof(float));
        memset(l, 0x00, N * sizeof(float));
        for (int j = 0; j < N; j++) {
            m[j] = -FLT_MAX;
        }

        // copy l and m to device
        float *dev_l, *dev_m;
        hipMalloc((void**)&dev_l, N * sizeof(float));
        hipMalloc((void**)&dev_m, N * sizeof(float));
        hipMemcpy(dev_l, l, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_m, m, N * sizeof(float), hipMemcpyHostToDevice);

        // kernel call
        flash_attention_kernel<<<blockNum, blockSize>>>(
            dev_Q + i * N * d, 
            dev_K + i * N * d, 
            dev_V + i * N * d, 
            dev_O + i * N * d,
            dev_l, dev_m, d
        );

        //free memory of l and m 
        free(l);
        free(m);
        hipFree(dev_l);
        hipFree(dev_m);
    }

    end = getTimeStamp();
    printf("(B, N, d): (%d, %d, %d)\n", B, N, d);
    printf("Time: %.3f seconds\n", end - start);

    // copy cuda memory back to host
    hipMemcpy(O, dev_O, B * N * d * sizeof(float), hipMemcpyDeviceToHost);

    output(argv[2]);

    // free memory
    hipFree(dev_Q);
    hipFree(dev_K);
    hipFree(dev_V);
    hipFree(dev_O);

    return 0;
}

void input(char *input_filename) {
    FILE *file = fopen(input_filename, "rb");

    fread(&B, sizeof(int), 1, file);
    fread(&N, sizeof(int), 1, file);
    fread(&d, sizeof(int), 1, file);

    Q = (float *)malloc(B * N * d * sizeof(float));
    K = (float *)malloc(B * N * d * sizeof(float));
    V = (float *)malloc(B * N * d * sizeof(float));
    O = (float *)malloc(B * N * d * sizeof(float));

    for (int i = 0; i < B; i++) {
        fread(Q + (i * N * d), sizeof(float), N * d, file);
        fread(K + (i * N * d), sizeof(float), N * d, file);
        fread(V + (i * N * d), sizeof(float), N * d, file);
    }
    memset(O, 0x00, B * N * d * sizeof(float));

    fclose(file);
}

void output(char *output_filename) {
    FILE *file = fopen(output_filename, "wb");

    fwrite(O, sizeof(float), B * N * d, file);

    free(Q);
    free(K);
    free(V);
    free(O);

    fclose(file);
}

__global__ void flash_attention_kernel(float* q, float* k, float* v, float* o, float* l, float* m, int d){
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int block_j = blockIdx.x;
    int block_i = blockIdx.y;
    int dim = d;
    int br = BLOCK_SIZE, bc = BLOCK_SIZE;
    
    // shared memory declaration 
    __shared__ float qi[BLOCK_SIZE * 64];
    __shared__ float kj[BLOCK_SIZE * 64];
    __shared__ float vj[BLOCK_SIZE * 64];
    __shared__ float oi[BLOCK_SIZE * 64];
    __shared__ float li[BLOCK_SIZE];
    __shared__ float mi[BLOCK_SIZE];

    __shared__ float sij[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float pij[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float mij[BLOCK_SIZE];
    __shared__ float lij[BLOCK_SIZE];

    __shared__ float mi_new[BLOCK_SIZE];
    __shared__ float li_new[BLOCK_SIZE];

    // init shared memory
    kj[thread_x * dim + thread_y] = k[(block_j * bc + thread_x) * dim + thread_y];
    vj[thread_x * dim + thread_y] = v[(block_j * bc + thread_x) * dim + thread_y];
    qi[thread_y * dim + thread_x] = q[(block_i * br + thread_y) * dim + thread_x];
    oi[thread_y * dim + thread_x] = o[(block_i * br + thread_y) * dim + thread_x];
    if(dim == 64){
        kj[thread_x * dim + (bc + thread_y)] = k[(block_j * bc + thread_x) * dim + (bc + thread_y)];
        vj[thread_x * dim + (bc + thread_y)] = v[(block_j * bc + thread_x) * dim + (bc + thread_y)];
        qi[thread_y * dim + (br + thread_x)] = q[(block_i * br + thread_y) * dim + (br + thread_x)];
        oi[thread_y * dim + (br + thread_x)] = o[(block_i * br + thread_y) * dim + (br + thread_x)];
    }
    if(thread_x == 0){ // only need one thread for each row to initial li and mi
        li[thread_y] = l[block_i * br + thread_y];
        mi[thread_y] = m[block_i * br + thread_y];
    }
    __syncthreads();

    /* QKDotAndScalar(sij, qi, kj, br, bc, 1.0/sqrt(d)) */
    sij[thread_y * bc + thread_x] = 0.0F;
    for (int t = 0; t < dim; t++)
    {
        sij[thread_y * bc + thread_x] += qi[thread_y * dim + t] * kj[thread_x * dim + t];
    }
    sij[thread_y * bc + thread_x] *= (1.0 / sqrtf(dim));
    __syncthreads();

    /* RowMax(mij, sij, br, bc) */
    if(thread_x == 0){ // only need one thread for each row to compute row max
        mij[thread_y] = sij[thread_y * bc];
        for (int j = 0; j < bc; j++)
        {
            mij[thread_y] = max(mij[thread_y], sij[thread_y * bc + j]);
        }
    }
    __syncthreads();

    /*  MinusMaxAndExp(pij, sij, mij, br, bc) */
    pij[thread_y * bc + thread_x] = expf(sij[thread_y * bc + thread_x] - mij[thread_y]);
    __syncthreads();

    /* RowSum(lij, pij, br, bc) */
    if(thread_x == 0){ // only need one thread for each row to sum up row
        lij[thread_y] = 0.0F;
        for (int j = 0; j < bc; j++)
        {
            lij[thread_y] += pij[thread_y * bc + j];
        }
    }
    __syncthreads();
    
    /* UpdateMiLiOi(mi, li, oi, mij, lij, pij, vj, br, bc) */
    if(thread_x == 0){ // only need one thread for each row to do this
        mi_new[thread_y] = max(mi[thread_y], mij[thread_y]);
        li_new[thread_y] = expf(mi[thread_y] - mi_new[thread_y]) * li[thread_y] 
                         + expf(mij[thread_y] - mi_new[thread_y]) * lij[thread_y];

        __syncthreads();

        for (int j = 0; j < dim; j++)
        {
            float pv = 0.0F;
            for (int t = 0; t < bc; t++)
            {
                pv += pij[thread_y * bc + t] * vj[t * dim + j];
            }
            oi[thread_y * dim + j] = (li[thread_y] * expf(mi[thread_y] - mi_new[thread_y]) * oi[thread_y * dim + j]
                                    + expf(mij[thread_y] - mi_new[thread_y]) * pv) / li_new[thread_y];
        }
    }
    __syncthreads();

    // copy memory back
    o[(block_i * br + thread_y) * dim + thread_x] = oi[thread_y * dim + thread_x];
    if(dim == 64) 
       o[(block_i * br + thread_y) * dim + (br + thread_x)] = oi[thread_y * dim + (br + thread_x)];
    if(thread_x == 0){
        l[block_i * br + thread_y] = li_new[thread_y];
        m[block_i * br + thread_y] = mi_new[thread_y];
    }
}